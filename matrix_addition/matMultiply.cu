#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

texture<int> texB;
texture<int, 2> tex2dB;
__constant__ int constB[16384];

__global__ void globalMultiplyKernel(int* dev_A, int* dev_B, int* dev_C, size_t matSize)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    while (x < matSize && y < matSize)
    {
        dev_C[x * matSize + y] = 0;
        for (size_t i = 0; i < matSize; i++)
            dev_C[x * matSize + y] += dev_A[x * matSize + i] * dev_B[i * matSize + y];
        
        x += gridDim.x * blockDim.x;
        y += gridDim.y * blockDim.y;
    }
}

__global__ void constantMultiplyKernel(int* dev_A, int* dev_C, size_t matSize)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    while (x < matSize && y < matSize)
    {
        dev_C[x * matSize + y] = 0;
        for (size_t i = 0; i < matSize; i++)
            dev_C[x * matSize + y] += dev_A[x * matSize + i] * constB[i * matSize + y];

        x += gridDim.x * blockDim.x;
        y += gridDim.y * blockDim.y;
    }
}

__global__ void texture1DMultiplyKernel(int* dev_A, int* dev_C, size_t matSize)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    while (x < matSize && y < matSize)
    {
        dev_C[x * matSize + y] = 0;
        for (size_t i = 0; i < matSize; i++)
            dev_C[x * matSize + y] += dev_A[x * matSize + i] * tex1Dfetch(texB, i * matSize + y);

        x += gridDim.x * blockDim.x;
        y += gridDim.y * blockDim.y;
    }
}

__global__ void texture2DMultiplyKernel(int* dev_A, int* dev_C, size_t matSize)
{
    
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    while (x < matSize && y < matSize)
    {
        dev_C[x * matSize + y] = 0;
        for (size_t i = 0; i < matSize; i++)
            dev_C[x * matSize + y] += dev_A[x * matSize + i] * tex2D(tex2dB, i, y);

        x += gridDim.x * blockDim.x;
        y += gridDim.y * blockDim.y;
    }
}

void globalMatMultiply(int* A, int* B, int* C, size_t matSize)
{
    int* dev_A, * dev_B, * dev_C;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_A, matSize * matSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_B, matSize * matSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_C, matSize * matSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    hipMemcpy(dev_A, A, matSize * matSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice Memcpy failed!");
        goto Error;
    }
    hipMemcpy(dev_B, B, matSize * matSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice hipMalloc failed!");
        goto Error;
    }

    dim3 threads(16, 16, 1);
    dim3 blocks((matSize + 15) / 16, (matSize + 15) / 16, 1);
    globalMultiplyKernel << <blocks, threads >> > (dev_A, dev_B, dev_C, matSize);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(C, dev_C, matSize * matSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DeviceToHost failed!");
        goto Error;
    }

Error:
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
}

void constantMatMultiply(int* A, int* B, int* C, size_t matSize)
{
    int* dev_A, * dev_C;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_A, matSize * matSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_C, matSize * matSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(constB), B, matSize * matSize * sizeof(int), 0, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Constant HostToDevice Memcpy failed!");
        goto Error;
    }
    hipMemcpy(dev_A, A, matSize * matSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice Memcpy failed!");
        goto Error;
    }

    dim3 threads(16, 16, 1);
    dim3 blocks((matSize + 15) / 16, (matSize + 15) / 16, 1);
    constantMultiplyKernel << <blocks, threads >> > (dev_A, dev_C, matSize);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(C, dev_C, matSize * matSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DeviceToHost failed!");
        goto Error;
    }

Error:
    hipFree(dev_A);
    hipFree(dev_C);
}

void texture1DMatMultiply(int* A, int* B, int* C, size_t matSize)
{
    int* dev_A, * dev_B, * dev_C;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_A, matSize * matSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_B, matSize * matSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_C, matSize * matSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    hipMemcpy(dev_A, A, matSize * matSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice Memcpy failed!");
        goto Error;
    }
    hipMemcpy(dev_B, B, matSize * matSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice Memcpy failed!");
        goto Error;
    }

    hipBindTexture(NULL, texB, dev_B, matSize * matSize * sizeof(int));

    dim3 threads(16, 16, 1);
    dim3 blocks((matSize + 15) / 16, (matSize + 15) / 16, 1);
    texture1DMultiplyKernel << <blocks, threads >> > (dev_A, dev_C, matSize);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(C, dev_C, matSize * matSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DeviceToHost failed!");
        goto Error;
    }

Error:
    hipUnbindTexture(texB);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
}

void texture2DMatMultiply(int* A, int* B, int* C, size_t matSize)
{
    int* dev_A, * dev_B, * dev_C;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_A, matSize * matSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    // For texture2D, we have to make sure that array is aligned to memory?
    size_t pitch;
    hipMallocPitch((void**)&dev_B, &pitch, matSize * sizeof(int), matSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocPitch failed!");
        goto Error;
    }
//   cudaStatus = hipMalloc((void**)&dev_B, matSize * matSize * sizeof(int));
//     if (cudaStatus != hipSuccess) {
//       fprintf(stderr, "hipMalloc failed!");
//       goto Error;
//    }
    cudaStatus = hipMalloc((void**)&dev_C, matSize * matSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    hipMemcpy(dev_A, A, matSize * matSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice Memcpy failed!");
        goto Error;
    }
    hipMemcpy2D(dev_B, pitch, B, matSize * sizeof(int), matSize * sizeof(int), matSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice2D failed!");
        goto Error;
    }
//    hipMemcpy(dev_B, B, matSize * matSize * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "HostToDevice Memcpy failed!");
//        goto Error;
//    }

    hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
    hipBindTexture2D(NULL, tex2dB, dev_B, desc, matSize, matSize, pitch);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "texture binding failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    dim3 threads(16, 16, 1);
    dim3 blocks((matSize + 15) / 16, (matSize + 15) / 16, 1);
    texture2DMultiplyKernel << <blocks, threads >> > (dev_A, dev_C, matSize);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(C, dev_C, matSize * matSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DeviceToHost failed!");
        goto Error;
    }

Error:
    hipUnbindTexture(tex2dB);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
}