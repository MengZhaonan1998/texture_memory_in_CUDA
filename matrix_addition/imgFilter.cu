#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "cudaheader.cuh"
#include <stdio.h>

texture<int, 2> texMask;
__constant__ int constMask[4096];

__global__ void globalConvKernel(int* dev_inputImg, int* dev_convMask, size_t imgSize, size_t maskSize)
{
    int x = maskSize / 2 + threadIdx.x + blockIdx.x * blockDim.x;
    int y = maskSize / 2 + threadIdx.y + blockIdx.y * blockDim.y;

    while (x < imgSize - maskSize / 2 && y < imgSize - maskSize / 2)
    {
        for(size_t i = 0; i < maskSize; i++)
            for (size_t j = 0; j < maskSize; j++)
            {
                dev_inputImg[x + y * blockDim.x * gridDim.x] += 
                    dev_inputImg[(x + i - maskSize / 2) + (y + j - maskSize / 2) * blockDim.x * gridDim.x] * dev_convMask[i * maskSize + j];
            }
        x += blockDim.x * gridDim.x;
        y += blockDim.y * gridDim.y;
    }
}

__global__ void textureConvKernel(int* dev_inputImg, size_t imgSize, size_t maskSize)
{
    int x = maskSize / 2 + threadIdx.x + blockIdx.x * blockDim.x;
    int y = maskSize / 2 + threadIdx.y + blockIdx.y * blockDim.y;

    while (x < imgSize - maskSize / 2 && y < imgSize - maskSize / 2)
    {
        for (size_t i = 0; i < maskSize; i++)
            for (size_t j = 0; j < maskSize; j++)
            {
                dev_inputImg[x + y * blockDim.x * gridDim.x] +=
                    dev_inputImg[(x + i - maskSize / 2) + (y + j - maskSize / 2) * blockDim.x * gridDim.x] * tex2D(texMask, i, j);
            }
        x += blockDim.x * gridDim.x;
        y += blockDim.y * gridDim.y;
    }
}

__global__ void constantConvKernel(int* dev_inputImg, size_t imgSize, size_t maskSize)
{
    int x = maskSize / 2 + threadIdx.x + blockIdx.x * blockDim.x;
    int y = maskSize / 2 + threadIdx.y + blockIdx.y * blockDim.y;

    while (x < imgSize - maskSize / 2 && y < imgSize - maskSize / 2)
    {
        for (size_t i = 0; i < maskSize; i++)
            for (size_t j = 0; j < maskSize; j++)
            {
                dev_inputImg[x + y * blockDim.x * gridDim.x] +=
                    dev_inputImg[(x + i - maskSize / 2) + (y + j - maskSize / 2) * blockDim.x * gridDim.x] * constMask[i * maskSize + j];
            }
        x += blockDim.x * gridDim.x;
        y += blockDim.y * gridDim.y;
    }
}

void  globalFiltering(int* inputImg, int* convMask, size_t imgSize, size_t maskSize)
{
    int* dev_inputImg;
    int* dev_convMask;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_inputImg, imgSize * imgSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_convMask, maskSize * maskSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_inputImg, inputImg, imgSize * imgSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_convMask, convMask, maskSize * maskSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    dim3 blocks((imgSize - maskSize + 31) / 32, (imgSize - maskSize + 31) / 32);
    dim3 threads(32, 32);

    globalConvKernel << <blocks, threads >> > (dev_inputImg, dev_convMask, imgSize, maskSize);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(inputImg, dev_inputImg, imgSize * imgSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_inputImg);
}

void  textureFiltering(int* inputImg, int* convMask, size_t imgSize, size_t maskSize)
{
    int* dev_inputImg;
    int* dev_convMask;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_inputImg, imgSize * imgSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // For texture2D, we have to make sure that array is aligned to memory?
    size_t pitch;
    cudaStatus = hipMallocPitch((void**)&dev_convMask, &pitch, maskSize * sizeof(int), maskSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocPitch failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_inputImg, inputImg, imgSize * imgSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy2D(dev_convMask, pitch, convMask, maskSize * sizeof(int), maskSize * sizeof(int), maskSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice2D failed!");
        goto Error;
    }
    hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
    hipBindTexture2D(NULL, texMask, dev_convMask, desc, maskSize, maskSize, pitch);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "texture binding failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    dim3 blocks((imgSize - maskSize + 31) / 32, (imgSize - maskSize + 31) / 32);
    dim3 threads(32, 32);

    textureConvKernel << <blocks, threads >> > (dev_inputImg, imgSize, maskSize);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(inputImg, dev_inputImg, imgSize * imgSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    hipUnbindTexture(texMask);

Error:
    hipFree(dev_inputImg);
}

void constantFiltering(int* inputImg, int* convMask, size_t imgSize, size_t maskSize)
{
    int* dev_inputImg;
    int* dev_convMask;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_inputImg, imgSize * imgSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_inputImg, inputImg, imgSize * imgSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(constMask), convMask, maskSize * maskSize * sizeof(int), 0, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Constant HostToDevice Memcpy failed!");
        goto Error;
    }
    
    dim3 blocks((imgSize - maskSize + 31) / 32, (imgSize - maskSize + 31) / 32);
    dim3 threads(32, 32);

    constantConvKernel << <blocks, threads >> > (dev_inputImg, imgSize, maskSize);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(inputImg, dev_inputImg, imgSize * imgSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_inputImg);
}