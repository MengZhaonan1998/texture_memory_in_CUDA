#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "cudaheader.cuh"
#include <stdio.h>

texture<int> texIn;
//__constant__ int constIn[16384];

__global__ void global_filterKernel(int* dev_outputImg, int* dev_inputImg, size_t imgSize, filterKernel fk)
{
    int x = 1+ threadIdx.x + blockIdx.x * blockDim.x;
    int y = 1+ threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    
    if (x < (imgSize - 1) && y < (imgSize - 1))
    {
        dev_outputImg[offset] =
            fk.top * dev_inputImg[x + (y - 1) * blockDim.x * gridDim.x] +
            fk.left * dev_inputImg[(x - 1) + y * blockDim.x * gridDim.x] +
            fk.center * dev_inputImg[x + y * blockDim.x * gridDim.x] +
            fk.right * dev_inputImg[(x + 1) + y * blockDim.x * gridDim.x] +
            fk.bottom * dev_inputImg[x + (y + 1) * blockDim.x * gridDim.x];
    }      
}

__global__ void texture_filterKernel(int* dev_outputImg, size_t imgSize, filterKernel fk)
{
    int x = 1 + threadIdx.x + blockIdx.x * blockDim.x;
    int y = 1 + threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    
    int left = offset - 1;
    int right = offset + 1;
    int top = offset - imgSize;
    int bottom = offset + imgSize;
    
    if (x < (imgSize - 1) && y < (imgSize - 1))
    {
        dev_outputImg[offset] =
            fk.top * tex1Dfetch(texIn, top) +
            fk.left * tex1Dfetch(texIn, left) +
            fk.center * tex1Dfetch(texIn, offset) +
            fk.right * tex1Dfetch(texIn, right) +
            fk.bottom * tex1Dfetch(texIn, bottom);
    }
}

void  textureAccess(int* outputImg, int* inputImg, size_t imgSize, filterKernel fk)
{
    int* dev_inputImg;
    int* dev_outputImg;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    hipMalloc((void**)&dev_inputImg, imgSize * imgSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    hipMalloc((void**)&dev_outputImg, imgSize * imgSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_inputImg, inputImg, imgSize * imgSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice hipMemcpy failed!");
        goto Error;
    }

    //hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
    //hipBindTexture2D(0, texIn, dev_inputImg, desc, imgSize, imgSize, sizeof(int) * imgSize);
    hipBindTexture(NULL, texIn, dev_inputImg, imgSize * imgSize * sizeof(int));

    dim3 blocks((imgSize+31) / 32, (imgSize + 31) / 32);
    dim3 threads(32, 32);
    texture_filterKernel << <blocks, threads >> > (dev_outputImg, imgSize, fk);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(outputImg, dev_outputImg, imgSize * imgSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DeviceToHost hipMemcpy failed!");
        goto Error;
    }

    Error:
    hipUnbindTexture(texIn);
    hipFree(dev_inputImg);
    hipFree(dev_outputImg);
}

void  globalAccess(int* outputImg, int* inputImg, size_t imgSize, filterKernel fk)
{
    int* dev_inputImg;
    int* dev_outputImg;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    hipMalloc((void**)&dev_inputImg, imgSize * imgSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    hipMalloc((void**)&dev_outputImg, imgSize * imgSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_inputImg, inputImg, imgSize * imgSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice hipMemcpy failed!");
        goto Error;
    }

    dim3 blocks((imgSize + 31) / 32, (imgSize + 31) / 32);
    dim3 threads(32, 32);
    global_filterKernel << <blocks, threads >> > (dev_outputImg, dev_inputImg, imgSize, fk);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    

    cudaStatus = hipMemcpy(outputImg, dev_outputImg, imgSize * imgSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DeviceToHost hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_inputImg);
    hipFree(dev_outputImg);
}

