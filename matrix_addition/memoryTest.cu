#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "cudaheader.cuh"
#include <stdio.h>

texture<int, 2> texIn;
//__constant__ int constIn[16384];

__global__ void global_filterKernel(int* dev_outputImg, int* dev_inputImg, size_t imgSize, filterKernel fk)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = (x + 1) + (y + 1) * blockDim.x * gridDim.x;
    if (x < (imgSize - 1) && y < (imgSize - 1))
    {
        dev_outputImg[offset] =
            fk.top * dev_inputImg[x + (y - 1) * blockDim.x * gridDim.x] +
            fk.left * dev_inputImg[(x - 1) + y * blockDim.x * gridDim.x] +
            fk.center * dev_inputImg[x + y * blockDim.x * gridDim.x] +
            fk.right * dev_inputImg[(x + 1) + y * blockDim.x * gridDim.x] +
            fk.bottom * dev_inputImg[x + (y + 1) * blockDim.x * gridDim.x];
    }      
}

__global__ void texture_filterKernel(int* dev_outputImg, size_t imgSize, filterKernel fk)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = (x + 1) + (y + 1) * blockDim.x * gridDim.x;
    if (x < (imgSize - 1) && y < (imgSize - 1))
    {
        dev_outputImg[offset] =
            fk.top * tex2D(texIn, x, y - 1) +
            fk.left * tex2D(texIn, x - 1, y) +
            fk.center * tex2D(texIn, x, y) +
            fk.right * tex2D(texIn, x + 1, y) +
            fk.bottom * tex2D(texIn, x, y + 1);
    }
}
/*
__global__ void const_filterKernel(int* d_outputImg, filterKernel fk)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    d_outputImg[offset] =
        fk.top_left * constIn[x + y * blockDim.x * gridDim.x] +
        fk.top * constIn[(x + 1) + y * blockDim.x * gridDim.x] +
        fk.top_right * constIn[(x + 2) + y * blockDim.x * gridDim.x] +
        fk.left * constIn[x + (y + 1) * blockDim.x * gridDim.x] +
        fk.center * constIn[(x + 1) + (y + 1) * blockDim.x * gridDim.x] +
        fk.right * constIn[(x + 2) + (y + 1) * blockDim.x * gridDim.x] +
        fk.bottom_left * constIn[x + (y + 2) * blockDim.x * gridDim.x] +
        fk.bottom * constIn[(x + 1) + (y + 2) * blockDim.x * gridDim.x] +
        fk.bottom_right * constIn[(x + 2) + (y + 2) * blockDim.x * gridDim.x];
}
*/


void  textureAccess(int* outputImg, int* inputImg, size_t imgSize, filterKernel fk)
{
    int* dev_inputImg;
    int* dev_outputImg;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    hipMalloc((void**)&dev_inputImg, imgSize * imgSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    hipMalloc((void**)&dev_outputImg, imgSize * imgSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_inputImg, inputImg, imgSize * imgSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
    hipBindTexture2D(NULL, texIn, dev_inputImg, desc, imgSize, imgSize, sizeof(int) * imgSize);

    dim3 blocks((imgSize+31) / 32, (imgSize + 31) / 32);
    dim3 threads(32, 32);
    texture_filterKernel << <blocks, threads >> > (dev_outputImg, imgSize, fk);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(outputImg, dev_outputImg, imgSize * imgSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Error:
    hipUnbindTexture(texIn);
    hipFree(dev_inputImg);
    hipFree(dev_outputImg);
}

void  globalAccess(int* outputImg, int* inputImg, size_t imgSize, filterKernel fk)
{
    int* dev_inputImg;
    int* dev_outputImg;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    hipMalloc((void**)&dev_inputImg, imgSize * imgSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    hipMalloc((void**)&dev_outputImg, imgSize * imgSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_inputImg, inputImg, imgSize * imgSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice hipMemcpy failed!");
        goto Error;
    }

    dim3 blocks((imgSize + 31) / 32, (imgSize + 31) / 32);
    dim3 threads(32, 32);
    global_filterKernel << <blocks, threads >> > (dev_outputImg, dev_inputImg, imgSize, fk);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(outputImg, dev_outputImg, imgSize * imgSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DeviceToHost hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_inputImg);
    hipFree(dev_outputImg);
}

/*
void constantAccess(int* outputImg, int* inputImg, size_t imgDim, filterKernel fk)
{
    int* d_outputImg;
    hipMemcpyToSymbol(HIP_SYMBOL("constIn"), inputImg, 16384 * sizeof(int), 0, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_outputImg, (imgDim - 2) * (imgDim - 2) * sizeof(int));

    dim3 blocks(imgDim / 32, imgDim / 32);
    dim3 threads(32, 32);

    const_filterKernel << <blocks, threads >> > (d_outputImg, fk);

    hipMemcpy(outputImg, d_outputImg, (imgDim - 2) * (imgDim - 2) * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_outputImg);
}
*/