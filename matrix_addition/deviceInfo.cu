﻿#include "hip/hip_runtime.h"

#include <stdio.h>

void GetDeviceInfo()
{
	hipError_t result;
	result = hipInit(0);
	hipDevice_t device;
	result = hipDeviceGet(&device, 0);

	int texture1Dwidth;
	int texture2Dwidth;
	int texture2Dheight;

	result = hipDeviceGetAttribute(&texture1Dwidth, hipDeviceAttributeMaxTexture1DWidth, device);
	printf("CU_DEVICE_ATTRIBUTE_MAXIMUM_TEXTURE1D_WIDTH = %d KBs\n", texture1Dwidth/1024);

	result = hipDeviceGetAttribute(&texture2Dwidth, hipDeviceAttributeMaxTexture2DWidth, device);
	printf("CU_DEVICE_ATTRIBUTE_MAXIMUM_TEXTURE2D_WIDTH = %d KBs\n", texture2Dwidth/1024);
	
	result = hipDeviceGetAttribute(&texture2Dheight, hipDeviceAttributeMaxTexture2DHeight, device);
	printf("CU_DEVICE_ATTRIBUTE_MAXIMUM_TEXTURE2D_HEIGHT = %d KBs\n", texture2Dheight/1024);
}