#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <stdio.h>

texture<int> texA;
texture<int> texB;

__constant__ int constA[8192];
__constant__ int constB[8192];

__global__ void globalAddKernel(int* dev_A, int* dev_B, int* dev_C, int arraySize)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    while (tidx < arraySize)
    {
        dev_C[tidx] = dev_A[tidx] + dev_B[tidx];
        tidx += blockDim.x * gridDim.x;
    }
}

__global__ void constantAddKernel(int* dev_C, int arraySize)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    while (tidx < arraySize)
    {
        dev_C[tidx] = constA[tidx] + constB[tidx];
        tidx += blockDim.x * gridDim.x;
    }
}

__global__ void textureAddKernel(int* dev_C, int arraySize)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    while (tidx < arraySize)
    {
        dev_C[tidx] = tex1Dfetch(texA, tidx) + tex1Dfetch(texB, tidx);
        tidx += blockDim.x * gridDim.x;
    }   
}

void globalArrayAdd(int* A, int* B, int* C, size_t arraySize)
{
    int* dev_A, *dev_B, *dev_C;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_A, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_B, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_C, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    hipMemcpy(dev_A, A, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice Memcpy failed!");
        goto Error;
    }
    hipMemcpy(dev_B, B, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice hipMalloc failed!");
        goto Error;
    }

    dim3 threads(128, 1, 1);
    dim3 blocks((arraySize + 127) / 128, 1, 1);
    globalAddKernel << <blocks, threads >> > (dev_A, dev_B, dev_C, arraySize);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(C, dev_C, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DeviceToHost failed!");
        goto Error;
    }

    Error:
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
}

void textureArrayAdd(int* A, int* B, int* C, size_t arraySize)
{
    int* dev_A, * dev_B, * dev_C;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_A, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_B, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_C, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    hipMemcpy(dev_A, A, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice Memcpy failed!");
        goto Error;
    }
    hipMemcpy(dev_B, B, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HostToDevice hipMalloc failed!");
        goto Error;
    }

    hipBindTexture(NULL, texA, dev_A, arraySize * sizeof(int));
    hipBindTexture(NULL, texB, dev_B, arraySize * sizeof(int));
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Texture binding failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    dim3 threads(128, 1, 1);
    dim3 blocks((arraySize + 127) / 128, 1, 1);
    textureAddKernel << <blocks, threads >> > (dev_C, arraySize);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(C, dev_C, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DeviceToHost failed!");
        goto Error;
    }

Error:
    hipUnbindTexture(texA);
    hipUnbindTexture(texB);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
}

void constantArrayAdd(int* A, int* B, int* C, size_t arraySize)
{
    int* dev_C;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    /*cudaStatus = hipMalloc((void**)&dev_A, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_B, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }*/
    cudaStatus = hipMalloc((void**)&dev_C, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(constA), A, 8192 * sizeof(int), 0, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Constant HostToDevice Memcpy failed!");
        goto Error;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(constB), B, 8192 * sizeof(int), 0, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Constant HostToDevice hipMalloc failed!");
        goto Error;
    }

    dim3 threads(128, 1, 1);
    dim3 blocks((arraySize + 127) / 128, 1, 1);
    constantAddKernel << <blocks, threads >> > (dev_C, arraySize);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(C, dev_C, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DeviceToHost failed!");
        goto Error;
    }

Error:
    hipFree(dev_C);
}
